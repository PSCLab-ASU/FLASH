#include "hip/hip_runtime.h"
//bondsKernelsGpu.cu
//Scott Grauer-Gray
//Bonds kernels to run on the GPU

#include "cfd.h"

__device__
inline void compute_velocity(const float density, const Float3 momentum, Float3* velocity){
  velocity->x = momentum.x / density;
  velocity->y = momentum.y / density;
  velocity->z = momentum.z / density;
}

__device__
inline float compute_speed_sqd(const Float3 velocity){
  return velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z;
}

__device__
inline float compute_pressure(const float density, const float density_energy, const float speed_sqd){
  return ((float)(GAMMA) - (float)(1.0f))*(density_energy - (float)(0.5f)*density*speed_sqd);
}
// sqrt is a device function
__device__
inline float compute_speed_of_sound(const float density, const float pressure){
  return sqrt((float)(GAMMA)*pressure/density);
}
  __device__ __host__
inline void compute_flux_contribution(const float density, 
    Float3 momentum, 
    const float density_energy, 
    const float pressure, 
    const Float3 velocity, 
    Float3* fc_momentum_x, 
    Float3* fc_momentum_y, 
    Float3* fc_momentum_z, 
    Float3* fc_density_energy)
{
  fc_momentum_x->x = velocity.x*momentum.x + pressure;
  fc_momentum_x->y = velocity.x*momentum.y;
  fc_momentum_x->z = velocity.x*momentum.z;


  fc_momentum_y->x = fc_momentum_x->y;
  fc_momentum_y->y = velocity.y*momentum.y + pressure;
  fc_momentum_y->z = velocity.y*momentum.z;

  fc_momentum_z->x = fc_momentum_x->z;
  fc_momentum_z->y = fc_momentum_y->z;
  fc_momentum_z->z = velocity.z*momentum.z + pressure;

  const float de_p = density_energy+pressure;
  fc_density_energy->x = velocity.x*de_p;
  fc_density_energy->y = velocity.y*de_p;
  fc_density_energy->z = velocity.z*de_p;
}

__global__ void init_buffer(int nelr, const float * val,  float *d)
{
  const int i = (blockDim.x*blockIdx.x + threadIdx.x);
  if (i < nelr) d[i] = val[0];
}

__global__ void init_variables(int nelr, const float* ff_variable, float * variables)
{
  const int i = (blockDim.x*blockIdx.x + threadIdx.x);
  for(int j = 0; j < NVAR; j++)
    variables[i + j*nelr] = ff_variable[j];
}


__global__ void compute_step_factor(const int nelr, 
    const float* variables, 
    const float* areas, 
    float* step_factors){

  const int i = (blockDim.x*blockIdx.x + threadIdx.x);
  if( i >= nelr) return;

  float density = variables[i + VAR_DENSITY*nelr];
  Float3 momentum;
  momentum.x = variables[i + (VAR_MOMENTUM+0)*nelr];
  momentum.y = variables[i + (VAR_MOMENTUM+1)*nelr];
  momentum.z = variables[i + (VAR_MOMENTUM+2)*nelr];

  float density_energy = variables[i + VAR_DENSITY_ENERGY*nelr];

  Float3 velocity;       compute_velocity(density, momentum, &velocity);
  float speed_sqd      = compute_speed_sqd(velocity);

  float pressure       = compute_pressure(density, density_energy, speed_sqd);
  float speed_of_sound = compute_speed_of_sound(density, pressure);
  step_factors[i] = (float)(0.5f) / (sqrt(areas[i]) * (sqrt(speed_sqd) + speed_of_sound));
}


__global__ void 
compute_flux(
    int nelr, //in
    int* elements_surrounding_elements, //in
    float* normals, //in
    float* variables, //in
    float* ff_variable, //in
    Float3* ff_flux_contribution_density_energy, //in
    Float3* ff_flux_contribution_momentum_x, //in
    Float3* ff_flux_contribution_momentum_y, //in 
    Float3* ff_flux_contribution_momentum_z, //in
    float* fluxes //out
    ){

  const int i = (blockDim.x*blockIdx.x + threadIdx.x);

  if( i >= nelr) return;
  const float smoothing_coefficient = (float)(0.2f);
  int j, nb;
  Float3 normal; 
  float normal_len;
  float factor;

  float density_i = variables[i + VAR_DENSITY*nelr];
  Float3 momentum_i;
  momentum_i.x = variables[i + (VAR_MOMENTUM+0)*nelr];
  momentum_i.y = variables[i + (VAR_MOMENTUM+1)*nelr];
  momentum_i.z = variables[i + (VAR_MOMENTUM+2)*nelr];

  float density_energy_i = variables[i + VAR_DENSITY_ENERGY*nelr];

  Float3 velocity_i;                     
  compute_velocity(density_i, momentum_i, &velocity_i);
  float speed_sqd_i                          = compute_speed_sqd(velocity_i);
  //float speed_sqd_i;
  //compute_speed_sqd(velocity_i, speed_sqd_i);
  float speed_i                              = sqrt(speed_sqd_i);
  float pressure_i                           = compute_pressure(density_i, density_energy_i, speed_sqd_i);
  float speed_of_sound_i                     = compute_speed_of_sound(density_i, pressure_i);
  Float3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z;
  Float3 flux_contribution_i_density_energy;  
  compute_flux_contribution(density_i, momentum_i, density_energy_i, pressure_i, velocity_i, 
      &flux_contribution_i_momentum_x, &flux_contribution_i_momentum_y, 
      &flux_contribution_i_momentum_z, &flux_contribution_i_density_energy);

  float flux_i_density = (float)(0.0f);
  Float3 flux_i_momentum;
  flux_i_momentum.x = (float)(0.0f);
  flux_i_momentum.y = (float)(0.0f);
  flux_i_momentum.z = (float)(0.0f);
  float flux_i_density_energy = (float)(0.0f);

  Float3 velocity_nb;
  float density_nb, density_energy_nb;
  Float3 momentum_nb;
  Float3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z;
  Float3 flux_contribution_nb_density_energy;  
  float speed_sqd_nb, speed_of_sound_nb, pressure_nb;

#pragma unroll
  for(j = 0; j < NNB; j++)
  {
    nb = elements_surrounding_elements[i + j*nelr];
    normal.x = normals[i + (j + 0*NNB)*nelr];
    normal.y = normals[i + (j + 1*NNB)*nelr];
    normal.z = normals[i + (j + 2*NNB)*nelr];
    normal_len = sqrt(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);

    if(nb >= 0)   // a legitimate neighbor
    {
      density_nb = variables[nb + VAR_DENSITY*nelr];
      momentum_nb.x = variables[nb + (VAR_MOMENTUM+0)*nelr];
      momentum_nb.y = variables[nb + (VAR_MOMENTUM+1)*nelr];
      momentum_nb.z = variables[nb + (VAR_MOMENTUM+2)*nelr];
      density_energy_nb = variables[nb + VAR_DENSITY_ENERGY*nelr];
      compute_velocity(density_nb, momentum_nb, &velocity_nb);
      speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
      pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
      speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
      compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, 
          &flux_contribution_nb_momentum_x, &flux_contribution_nb_momentum_y, &flux_contribution_nb_momentum_z, 
          &flux_contribution_nb_density_energy);

      // artificial viscosity
      factor = -normal_len*smoothing_coefficient*(float)(0.5f)*(speed_i + sqrt(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
      flux_i_density += factor*(density_i-density_nb);
      flux_i_density_energy += factor*(density_energy_i-density_energy_nb);
      flux_i_momentum.x += factor*(momentum_i.x-momentum_nb.x);
      flux_i_momentum.y += factor*(momentum_i.y-momentum_nb.y);
      flux_i_momentum.z += factor*(momentum_i.z-momentum_nb.z);

      // accumulate cell-centered fluxes
      factor = (float)(0.5f)*normal.x;
      flux_i_density += factor*(momentum_nb.x+momentum_i.x);
      flux_i_density_energy += factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x);
      flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x);
      flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x);
      flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x);

      factor = (float)(0.5f)*normal.y;
      flux_i_density += factor*(momentum_nb.y+momentum_i.y);
      flux_i_density_energy += factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y);
      flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y);
      flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y);
      flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y);

      factor = (float)(0.5f)*normal.z;
      flux_i_density += factor*(momentum_nb.z+momentum_i.z);
      flux_i_density_energy += factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z);
      flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z);
      flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z);
      flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z);
    }
    else if(nb == -1)  // a wing boundary
    {
      flux_i_momentum.x += normal.x*pressure_i;
      flux_i_momentum.y += normal.y*pressure_i;
      flux_i_momentum.z += normal.z*pressure_i;
    }
    else if(nb == -2) // a far field boundary
    {
      factor = (float)(0.5f)*normal.x;
      flux_i_density += factor*(ff_variable[VAR_MOMENTUM+0]+momentum_i.x);
      flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].x+flux_contribution_i_density_energy.x);
      flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].x + flux_contribution_i_momentum_x.x);
      flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].x + flux_contribution_i_momentum_y.x);
      flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].x + flux_contribution_i_momentum_z.x);

      factor = (float)(0.5f)*normal.y;
      flux_i_density += factor*(ff_variable[VAR_MOMENTUM+1]+momentum_i.y);
      flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].y+flux_contribution_i_density_energy.y);
      flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].y + flux_contribution_i_momentum_x.y);
      flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].y + flux_contribution_i_momentum_y.y);
      flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].y + flux_contribution_i_momentum_z.y);

      factor = (float)(0.5f)*normal.z;
      flux_i_density += factor*(ff_variable[VAR_MOMENTUM+2]+momentum_i.z);
      flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].z+flux_contribution_i_density_energy.z);
      flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].z + flux_contribution_i_momentum_x.z);
      flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].z + flux_contribution_i_momentum_y.z);
      flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].z + flux_contribution_i_momentum_z.z);

    }
  }

  fluxes[i + VAR_DENSITY*nelr] = flux_i_density;
  fluxes[i + (VAR_MOMENTUM+0)*nelr] = flux_i_momentum.x;
  fluxes[i + (VAR_MOMENTUM+1)*nelr] = flux_i_momentum.y;
  fluxes[i + (VAR_MOMENTUM+2)*nelr] = flux_i_momentum.z;
  fluxes[i + VAR_DENSITY_ENERGY*nelr] = flux_i_density_energy;

}

__global__ void 
time_step(int j, int nelr, 
    const float* old_variables, 
    const float* step_factors, 
    const float* fluxes,
    float * variables) {

  const int i = (blockDim.x*blockIdx.x + threadIdx.x);
  if( i >= nelr) return;

  float factor = step_factors[i]/(float)(RK+1-j);

  variables[i + VAR_DENSITY*nelr] = old_variables[i + VAR_DENSITY*nelr] + factor*fluxes[i + VAR_DENSITY*nelr];
  variables[i + VAR_DENSITY_ENERGY*nelr] = old_variables[i + VAR_DENSITY_ENERGY*nelr] + factor*fluxes[i + VAR_DENSITY_ENERGY*nelr];
  variables[i + (VAR_MOMENTUM+0)*nelr] = old_variables[i + (VAR_MOMENTUM+0)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+0)*nelr];
  variables[i + (VAR_MOMENTUM+1)*nelr] = old_variables[i + (VAR_MOMENTUM+1)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+1)*nelr];  
  variables[i + (VAR_MOMENTUM+2)*nelr] = old_variables[i + (VAR_MOMENTUM+2)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+2)*nelr];  
}

